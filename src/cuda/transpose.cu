#include "hip/hip_runtime.h"
/*
 * @Author: lugy lugengyou@github.com
 * @Date: 2024-09-23 23:31:04
 * @FilePath: /lugy_hpc_libs/src/cuda/transpose.cu
 * @LastEditTime: 2024-09-24 00:27:41
 * @Description: transpose.cu
 */
#include "transpose.h"
#include <stdio.h>
#include <hip/hip_runtime.h>


static __global__ void transpose_kernel_v0(int *a, int *b, int srcN, int srcM) {
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.x * blockDim.x + threadIdx.x;

    if (r < srcN && c < srcM) {
        b[c * srcN + r] = a[r * srcM + c];
    }
}


void lugy::transpose_cuda(int *a, int *b, int srcN, int srcM, int version) {
    
    int *d_a, *d_b;
    
    hipMalloc((void **)&d_a, srcN * srcM * sizeof(int));
    hipMalloc((void **)&d_b, srcN * srcM * sizeof(int));
    hipMemcpy(d_a, a, srcN * srcM * sizeof(int), hipMemcpyHostToDevice);

    if (version == 0) {
        printf("cuda in transpose_kernel_v0\n");
        dim3 block(32, 32);
        dim3 grid((srcM + block.x - 1) / block.x, (srcN + block.y - 1) / block.y);            
        transpose_kernel_v0<<<grid, block>>>(d_a, d_b, srcN, srcM);
    }
    else {
        printf("version not found\n");
          
    }

    hipMemcpy(b, d_b, srcN * srcM * sizeof(int), hipMemcpyDeviceToHost);
    
    hipFree(d_a);
    hipFree(d_b);
}

