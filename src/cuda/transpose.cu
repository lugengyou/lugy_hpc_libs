#include "hip/hip_runtime.h"
/*
 * @Author: lugy lugengyou@github.com
 * @Date: 2024-09-23 23:31:04
 * @FilePath: /lugy_hpc_libs/src/cuda/transpose.cu
 * @LastEditTime: 2024-10-23 15:36:42
 * @Description: transpose.cu
 */
#include "transpose.h"
#include <stdio.h>
#include <hip/hip_runtime.h>


static __global__ void transpose_kernel_v0(int *a, int *b, int srcN, int srcM) {
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.x * blockDim.x + threadIdx.x;

    if (r < srcN && c < srcM) {
        b[c * srcN + r] = a[r * srcM + c];
    }
}

static __global__ void transpose_kernel_v1(int *a, int *b, int srcN, int srcM) {
    __shared__ int sdata[32][32];

    size_t y = threadIdx.y + blockIdx.y * blockDim.y; // 行
    size_t x = threadIdx.x + blockIdx.x * blockDim.x; // 列

    size_t src_addr = y * srcM + x;
    
    size_t share_idx = threadIdx.y * blockDim.x + threadIdx.x;

    // 计算共享内存转置对应的行列
    size_t row = share_idx / blockDim.y;
    size_t col = share_idx % blockDim.y;

    // 转置后的目标矩阵行列
    size_t trans_y = blockIdx.x * blockDim.x + row;
    size_t trans_x = blockIdx.y * blockDim.y + col; // trans_x 是连续增加的

    size_t dst_addr = trans_y * srcN + trans_x;

    if (y < srcN && x < srcM) {
        // 加载数据到共享内存
        sdata[threadIdx.y][threadIdx.x] = a[src_addr];
        __syncthreads();

        // 共享内存按列连续读取，转置按行连续存储     
        b[dst_addr] = sdata[col][row]; 
    }
}


void lugy::transpose_cuda(int *a, int *b, int srcN, int srcM, int version) {
    
    int *d_a, *d_b;
    
    hipMalloc((void **)&d_a, srcN * srcM * sizeof(int));
    hipMalloc((void **)&d_b, srcN * srcM * sizeof(int));
    hipMemcpy(d_a, a, srcN * srcM * sizeof(int), hipMemcpyHostToDevice);

    if (version == 0) {
        printf("cuda in transpose_kernel_v0\n");
        dim3 block(32, 32);
        dim3 grid((srcM + block.x - 1) / block.x, (srcN + block.y - 1) / block.y);            
        transpose_kernel_v0<<<grid, block>>>(d_a, d_b, srcN, srcM);
    }
    else if (version == 1) {
        printf("cuda in transpose_kernel_v1\n");
        dim3 block(32, 32);
        dim3 grid((srcM + block.x - 1) / block.x, (srcN + block.y - 1) / block.y);
        transpose_kernel_v1<<<grid, block>>>(d_a, d_b, srcN, srcM);
    }
    else {
        printf("version not found\n");
          
    }

    hipMemcpy(b, d_b, srcN * srcM * sizeof(int), hipMemcpyDeviceToHost);
    
    hipFree(d_a);
    hipFree(d_b);
}

